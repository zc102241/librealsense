
#include <hip/hip_runtime.h>
//This file is partly based on Mehran Maghoumi's work: https://github.com/Maghoumi/culibrealsense

#ifdef RS2_USE_CUDA

#include "cuda-conversion.cuh"
#include <iostream>
#include <iomanip>

/*
// conversion to Y8 is currently not available in the API
__global__ void kernel_unpack_yuy2_y8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= superPixCount)
		return;

	int idx = i * 4;
	
	dst[idx] = src[idx];
	dst[idx + 1] = src[idx + 2];
	dst[idx + 2] = src[idx + 4];
	dst[idx + 3] = src[idx + 6];
	dst[idx + 4] = src[idx + 8];
	dst[idx + 5] = src[idx + 10];
	dst[idx + 6] = src[idx + 12];
    dst[idx + 7] = src[idx + 14];
    dst[idx + 8] = src[idx + 16];
    dst[idx + 9] = src[idx + 18];
    dst[idx + 10] = src[idx + 20];
    dst[idx + 11] = src[idx + 22];
    dst[idx + 12] = src[idx + 24];
    dst[idx + 13] = src[idx + 26];
    dst[idx + 14] = src[idx + 28];
    dst[idx + 15] = src[idx + 30];
}
*/

__global__ void kernel_unpack_yuy2_y16_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
	if (i >= superPixCount)
		return;
		
	for (; i < superPixCount; i += stride) {

        int idx = i * 4;

        dst[idx] = 0;
        dst[idx + 1] = src[idx + 0];
        dst[idx + 2] = 0;
        dst[idx + 3] = src[idx + 2];
    }
}


__global__ void kernel_unpack_yuy2_rgb8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
	if (i >= superPixCount)
		return;
		
	for (; i < superPixCount; i += stride) {

	    int idx = i * 4;

	    uint8_t y0 = src[idx];
	    uint8_t u0 = src[idx + 1];
	    uint8_t y1 = src[idx + 2];
	    uint8_t v0 = src[idx + 3];

	    int16_t c = y0 - 16;
	    int16_t d = u0 - 128;
	    int16_t e = v0 - 128;

	    int32_t t;
    #define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)
        
	    int odx = i * 6;

	    dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);

	    c = y1 - 16;

	    dst[odx + 3] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx + 5] = clamp((298 * c + 516 * d + 128) >> 8);

    #undef clamp

    }
}

__global__ void kernel_unpack_yuy2_bgr8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

	if (i >= superPixCount)
		return;
		
	for (; i < superPixCount; i += stride) {

	    int idx = i * 4;

	    uint8_t y0 = src[idx];
	    uint8_t u0 = src[idx + 1];
	    uint8_t y1 = src[idx + 2];
	    uint8_t v0 = src[idx + 3];

	    int16_t c = y0 - 16;
	    int16_t d = u0 - 128;
	    int16_t e = v0 - 128;

	    int32_t t;
    #define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

	    int odx = i * 6;

	    dst[odx + 2] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx    ] = clamp((298 * c + 516 * d + 128) >> 8);

	    c = y1 - 16;

	    dst[odx + 5] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx + 3] = clamp((298 * c + 516 * d + 128) >> 8);

    #undef clamp
    }
}


__global__ void kernel_unpack_yuy2_rgba8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	if (i >= superPixCount)
		return;
		
	for (; i < superPixCount; i += stride) {

	    int idx = i * 4;

	    uint8_t y0 = src[idx];
	    uint8_t u0 = src[idx + 1];
	    uint8_t y1 = src[idx + 2];
	    uint8_t v0 = src[idx + 3];

	    int16_t c = y0 - 16;
	    int16_t d = u0 - 128;
	    int16_t e = v0 - 128;

	    int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

	    int odx = i * 8;

	    dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);
	    dst[odx + 3] = 255;

	    c = y1 - 16;

	    dst[odx + 4] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 5] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx + 6] = clamp((298 * c + 516 * d + 128) >> 8);
	    dst[odx + 7] = 255;

#undef clamp
    }
}

__global__ void kernel_unpack_yuy2_bgra8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	if (i >= superPixCount)
		return;
		
	for (; i < superPixCount; i += stride) {
	
	    int idx = i * 4;

	    uint8_t y0 = src[idx];
	    uint8_t u0 = src[idx + 1];
	    uint8_t y1 = src[idx + 2];
	    uint8_t v0 = src[idx + 3];

	    int16_t c = y0 - 16;
	    int16_t d = u0 - 128;
	    int16_t e = v0 - 128;

	    int32_t t;
	
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

	    int odx = i * 8;

        dst[odx + 3] = 255;
	    dst[odx + 2] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx    ] = clamp((298 * c + 516 * d + 128) >> 8);

	    c = y1 - 16;

        dst[odx + 7] = 255;
	    dst[odx + 6] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 5] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx + 4] = clamp((298 * c + 516 * d + 128) >> 8);

#undef clamp
 }
}


void rscuda::unpack_yuy2_cuda_helper(const uint8_t* src, uint8_t* dst, int n, rs2_format format) 
{
/*    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    cudaEventRecord(start); */
    
	// How many super pixels do we have?
	int superPix = n / 2;
	uint8_t *devSrc = 0;
	uint8_t *devDst = 0;
	
	cudaError_t result = cudaMalloc(&devSrc, superPix * sizeof(uint8_t) * 4);
	assert(result == cudaSuccess);
	
	result = cudaMemcpy(devSrc, src, superPix * sizeof(uint8_t) * 4, cudaMemcpyHostToDevice);
	assert(result == cudaSuccess);
	
	int numBlocks = superPix / RS2_CUDA_THREADS_PER_BLOCK;
	int size;
	
	switch (format)
	{
	// conversion to Y8 is currently not available in the API
	/*	case RS2_FORMAT_Y8:
	    size = 1;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_y8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, superPix);
		break;
	*/
	case RS2_FORMAT_Y16:
		size = 2;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_y16_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, superPix);
		break;
	case RS2_FORMAT_RGB8:
	    size = 3;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_rgb8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, superPix);
		break;
	case RS2_FORMAT_BGR8:
	    size = 3;
        result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_bgr8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, superPix);
    	break;
	case RS2_FORMAT_RGBA8:
		size = 4;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_rgba8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, superPix);
		break;
	case RS2_FORMAT_BGRA8:
        size = 4;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_bgra8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, superPix);
		break;
	default:
		assert(false);
	}


	result = cudaGetLastError();
	assert(result == cudaSuccess);

	result = cudaMemcpy(dst, devDst, n * sizeof(uint8_t) * size, cudaMemcpyDeviceToHost);
	assert(result == cudaSuccess);
	
	cudaFree(devSrc);
	cudaFree(devDst);

/*	cudaEventRecord(stop);	
	cudaEventSynchronize(stop);
    float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << "\n"; */
}


__global__ void kernel_split_frame_y8_y8_from_y8i_cuda(uint8_t* a, uint8_t* b, int count, const rscuda::y8i_pixel * source)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= count)
		return;

    a[i] = source[i].l;
    b[i] = source[i].r;
}

void rscuda::y8_y8_from_y8i_cuda_helper(uint8_t* const dest[], int count, const rscuda::y8i_pixel * source)
{
/*    cudaEvent_t start, stop;
	cudaEventCreate(&start);
    cudaEventCreate(&stop);
	cudaEventRecord(start); */

    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    uint8_t* a = dest[0];
    uint8_t* b = dest[1];
    
    rscuda::y8i_pixel *devSrc = 0;
    uint8_t *devDst1 = 0; // for dest[0]
    uint8_t *devDst2 = 0; // for dest[1]
    
    cudaError_t result = cudaMalloc(&devSrc, count * sizeof(rscuda::y8i_pixel));
    assert(result == cudaSuccess);

    result = cudaMemcpy(devSrc, source, count * sizeof(rscuda::y8i_pixel), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    result = cudaMalloc(&devDst1, count * sizeof(uint8_t));
    assert(result == cudaSuccess);

    result = cudaMalloc(&devDst2, count * sizeof(uint8_t));
    assert(result == cudaSuccess);
    
    kernel_split_frame_y8_y8_from_y8i_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devDst1, devDst2, count, devSrc);

    result = cudaGetLastError();
    assert(result == cudaSuccess);
    
    result = cudaMemcpy(a, devDst1, count * sizeof(uint8_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
    result = cudaMemcpy(b, devDst2, count * sizeof(uint8_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);

    cudaFree(devSrc);
    cudaFree(devDst1);
    cudaFree(devDst2);
    
/*    cudaEventRecord(stop);
	cudaEventSynchronize(stop);
	float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << std::endl; */
}

__global__ void kernel_split_frame_y16_y16_from_y12i_cuda(uint16_t* a, uint16_t* b, int count, const rscuda::y12i_pixel * source)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= count)
		return;

    a[i] = source[i].l() << 6 | source[i].l() >> 4;
    b[i] = source[i].r() << 6 | source[i].r() >> 4;
}


void rscuda::y16_y16_from_y12i_10_cuda_helper(uint8_t* const dest[], int count, const rscuda::y12i_pixel * source)
{
/*
    cudaEvent_t start, stop;
	cudaEventCreate(&start);
    cudaEventCreate(&stop);
	cudaEventRecord(start); */
    
    source =  reinterpret_cast<const y12i_pixel*>(source);

    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    uint16_t* a = reinterpret_cast<uint16_t*>(dest[0]);
    uint16_t* b = reinterpret_cast<uint16_t*>(dest[1]);
    
    rscuda::y12i_pixel *devSrc = 0;
    uint16_t *devDst1 = 0; // for dest[0]
    uint16_t *devDst2 = 0; // for dest[1]
    
    cudaError_t result = cudaMalloc(&devSrc, count * sizeof(rscuda::y12i_pixel));
    assert(result == cudaSuccess);

    result = cudaMemcpy(devSrc, source, count * sizeof(rscuda::y12i_pixel), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    result = cudaMalloc(&devDst1, count * sizeof(uint16_t));
    assert(result == cudaSuccess);

    result = cudaMalloc(&devDst2, count * sizeof(uint16_t));
    assert(result == cudaSuccess);
    
    kernel_split_frame_y16_y16_from_y12i_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devDst1, devDst2, count, devSrc);

    result = cudaGetLastError();
    assert(result == cudaSuccess);
    
    result = cudaMemcpy(a, devDst1, count * sizeof(uint16_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
    result = cudaMemcpy(b, devDst2, count * sizeof(uint16_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);

    cudaFree(devSrc);
    cudaFree(devDst1);
    cudaFree(devDst2);
    
    /*
	cudaEventRecord(stop);
	cudaEventSynchronize(stop);
    float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << std::endl;
    */
}


__global__ void kernel_z16_y8_from_sr300_inzi_cuda (const uint16_t* source, uint8_t* const dest, int count)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= count)
		return;

    dest[i] = source[i] >> 2;
}

void rscuda::unpack_z16_y8_from_sr300_inzi_cuda (uint8_t * const dest, const uint16_t * source, int count) 
{
/*  cudaEvent_t start, stop;
	cudaEventCreate(&start);
    cudaEventCreate(&stop);
	cudaEventRecord(start); */
	
    uint16_t *devSrc = 0;
    uint8_t *devDst = 0;
     
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    
    cudaError_t result = cudaMalloc(&devSrc, count * sizeof(uint16_t));
    assert(result == cudaSuccess);

    result = cudaMemcpy(devSrc, source, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);
    
    result = cudaMalloc(&devDst, count * sizeof(uint8_t));
    assert(result == cudaSuccess);

    kernel_z16_y8_from_sr300_inzi_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, count);
    
    result = cudaMemcpy(dest, devDst, count * sizeof(uint8_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
    
    cudaFree(devSrc);
    cudaFree(devDst);
    
/*  cudaEventRecord(stop);
   	cudaEventSynchronize(stop);
    float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << std::endl; */
}

__global__ void kernel_z16_y16_from_sr300_inzi_cuda (uint16_t* const source, uint16_t* const dest, int count)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= count)
		return;

    dest[i] = source[i] << 6;
}

void rscuda::unpack_z16_y16_from_sr300_inzi_cuda(uint16_t * const dest, const uint16_t * source, int count) 
{
/*  cudaEvent_t start, stop;
	cudaEventCreate(&start);
    cudaEventCreate(&stop);
	cudaEventRecord(start); */
	
    uint16_t *devSrc = 0;
    uint16_t *devDst = 0;
     
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    
    cudaError_t result = cudaMalloc(&devSrc, count * sizeof(uint16_t));
    assert(result == cudaSuccess);

    result = cudaMemcpy(devSrc, source, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);
    
    result = cudaMalloc(&devDst, count * sizeof(uint16_t));
    assert(result == cudaSuccess);

    kernel_z16_y16_from_sr300_inzi_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, count);
    
    result = cudaMemcpy(dest, devDst, count * sizeof(uint16_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
    
    cudaFree(devSrc);
    cudaFree(devDst);
    
        	
/*	cudaEventRecord(stop);
	cudaEventSynchronize(stop);
    float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds << std::endl; */
}

#endif
